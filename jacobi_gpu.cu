#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h> 


void init(int N, double delta, double *U, double *U_old, double *F) {

	int temp_N = N + 2; //the boundries
	// Declare relative coordinates
	double x = -1.0;
	double y = -1.0;
	double x_lower = 0.0;
	double x_upper = 1.0 / 3.0;
	double y_lower = -2.0 / 3.0;
	double y_upper = -1.0 / 3.0;
	int i, j;
	for (i = 0; i < temp_N; i++)
	{
		for (j = 0; j < temp_N; j++)
		{
            	F[i * (temp_N)+j] = 0.0;
            	U[i * (temp_N)+j] = 0.0;
            	U_old[i * (temp_N)+j] = 0.0;
		    // Place radiator for F in the right place
		    if (x <= x_upper && x >= x_lower && y <= y_upper && y >= y_lower)
		    {
			// Set radiator value to 200 degrees
			F[i * temp_N + j] = 200.0;
		    }
		    // Place temperature for walls
		    if (i == (temp_N - 1) || i == 0 || j == (temp_N - 1))
		    {
		        // Set temperature to 20 degrees for 3 of the walls
		        U[i *(temp_N) + j]= 20.0;
		        U_old[i * (temp_N) + j] = 20.0;
		    }	
		    // Move relative coordinates by one unit of grid spacing
		    y += delta;
		}
	// Move relative coordinates by one unit of grid spacing
	x += delta;
	y = -1.0;
	}
}

__global__ void jacobi(int N, double delta2, double *U, double *U_old, double *F) {
        int new_N = (N+2);  
	int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int j;
	for (j = 1; j < new_N-1; j++)
	    {
	        // Calculate new value from surrounding points
	        U_old[i * new_N + j] = (U[i * new_N+ (j-1)] + U[i * new_N + (j+1)] + U[(i-1) * new_N + j] + U[(i + 1) * new_N + j] + (delta2 * F[i * new_N + j])) * 0.25;
	    }
	__syncthreads();
}

void print_matrix(int N, double *M)
{
	int temp_N = N + 2;
	int i, j;
	for (i = temp_N - 1; i >= 0; i--)
	{
	for (j = 0; j < temp_N; j++)
	{
	    // Swap indecies to show correct x and y-axes
	    printf("%.2f\t", M[j * temp_N + i]);
	}
	printf("\n");
	}
}

int main() {

	int N = 32;
	int k = 1000;
	int size = (N+2) * (N+2) * sizeof(double);
	double delta = 2.0 / ((double) N - 1.0);
	double delta2 = delta * delta;
	
	dim3 DimBlock(1);
	dim3 DimGrid(N);
	
	double *U_dev;
	double *U_old_dev;
	double *F_dev;
	double *temp;
	double *U_host;
	double *U_old_host;
	double *F_host;
	
	//alloctating memory on host
	U_host = (double *) malloc(size);
	U_old_host = (double *) malloc(size);
	F_host = (double *) malloc(size);
	
	//initializing the arrays
	init(N, delta, U_host, U_old_host, F_host);
	
	//allocating memory on device
	checkCudaErrors(hipMalloc((void**) &U_dev,size));
	checkCudaErrors(hipMalloc((void**) &U_old_dev,size));
	checkCudaErrors(hipMalloc((void**) &F_dev, size));
	
	//copying memory from CPU to GPU
	checkCudaErrors(hipMemcpy(U_dev, U_host, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(U_old_dev, U_old_host, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(F_dev, F_host, size, hipMemcpyHostToDevice));
	int h;
	for(h = 0; h < k; h++) {
	jacobi<<<DimGrid, DimBlock>>>(N, delta2, U_dev, U_old_dev, F_dev);
		//swapping pointers
        	temp = U_dev;
                U_dev = U_old_dev;
                U_old_dev = temp;
	}
	checkCudaErrors(hipMemcpy(U_host, U_dev, size, hipMemcpyDeviceToHost));
	
	print_matrix(N, U_host);
	//freeing the memory in the end
	free(U_host);
	free(U_old_host);
	free(F_host);
	checkCudaErrors(hipFree(U_dev));
	checkCudaErrors(hipFree(U_old_dev));
	checkCudaErrors(hipFree(F_dev));
	return 0;
}
