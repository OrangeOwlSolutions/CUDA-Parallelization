#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h> 


void init(int N, double delta, double *U0, double *U_old0, double *U1, double *U_old1, double *F) {

	int temp_N = N + 2; //the boundries
	int temp_N_half = temp_N/2;
	// Declare relative coordinates
	double x = -1.0;
	double y = -1.0;
	double x_lower = 0.0;
	double x_upper = 1.0 / 3.0;
	double y_lower = -2.0 / 3.0;
	double y_upper = -1.0 / 3.0;
	int i, j;
	for (i = 0; i < temp_N; i++)
	{
		for (j = 0; j < temp_N; j++)
		{
            	F[i * (temp_N)+j] = 0.0;
		if(i >= temp_N_half) {
			U1[(i-temp_N_half) *(temp_N) + j]= 0.0;
			U_old1[(i-temp_N_half) * (temp_N) + j] = 0.0;
		}else {
			U0[i *(temp_N) + j]= 0.0;
			U_old0[i * (temp_N) + j] = 0.0;
		}
		    // Place radiator for F in the right place
		    if (x <= x_upper && x >= x_lower && y <= y_upper && y >= y_lower)
		    {
			// Set radiator value to 200 degrees
			F[i * temp_N + j] = 200.0;
		    }
		    // Place temperature for walls
		    if (i == (temp_N - 1) || i == 0 || j == (temp_N - 1))
		    {	
			if(i >= temp_N_half) {
				// Set temperature to 20 degrees for 3 of the walls
				U1[(i-temp_N_half) *(temp_N) + j]= 20.0;
				U_old1[(i-temp_N_half) * (temp_N) + j] = 20.0;
			}else {
				// Set temperature to 20 degrees for 3 of the walls
				U0[i *(temp_N) + j]= 20.0;
				U_old0[i * (temp_N) + j] = 20.0;
			}
		    }	
		    // Move relative coordinates by one unit of grid spacing
		    y += delta;
			
		}
	// Move relative coordinates by one unit of grid spacing
	x += delta;
	y = -1.0;
	
	}
	
}

__global__ void jacobi(int N, int temp_N_half, double delta2, double *U, double *U_old, double *U_other, double *F, int device) {  
	//printf("hej%d\n",temp_N_half);	
	int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int j;
	if(device) {
		i = i-1;
	}
	if(i <= temp_N_half-1) {
		for (j = 1; j < N-1; j++)
		    {
			// Calculate new value from surrounding points
			if(!device && (i == temp_N_half-1)) {
				U_old[i * N + j] = (U[i * N + (j-1)] + U[i * N + (j+1)] + U[(i-1) * N + j] + U_other[/*(i) * N +*/1 + j] + (delta2 * F[i * N + j])) * 0.25;
			}else if(device && !i) {
				U_old[i * N + j] = (U[i * N + (j-1)] + U[i * N + (j+1)] + U_other[(temp_N_half-1) * N + j] + U[(i+1) * N + j] + (delta2 * F[i * N + j])) * 0.25;			
			} else {
				U_old[i * N + j] = (U[i * N + (j-1)] + U[i * N + (j+1)] + U[(i-1) * N + j] + U[(i+1) * N + j] + (delta2 * F[i * N + j])) * 0.25;
			}
		    }
	}
	__syncthreads();
}

void print_matrix(int N, double *M0, double *M1)
{
	int temp_N = N + 2;
	int temp_N_half = temp_N/2;
	int i, j;
	for (i = 0; i < temp_N; i++)
	{
	for (j = 0; j < temp_N; j++)
	{
	    // Swap indecies to show correct x and y-axes
	    	if(i >= temp_N_half) {
	    		printf("%.2f\t", M1[(i-temp_N_half) * temp_N + j]);
		} else {
			printf("%.2f\t", M0[i * temp_N + j]);
		}
	}
	printf("\n");
	}
}

int main() {

	int N = 16;
	int k = 10000;
	int bs = 4;
	int N_half = N/2;
	int temp_N = N+2;
	int size = temp_N * temp_N * sizeof(double);
	double delta = 2.0 / ((double) N - 1.0);
	double delta2 = delta * delta;
	
	dim3 DimBlock(bs);
	dim3 DimGrid((N_half+DimBlock.x-1)/DimBlock.x);
	
	double *U_dev0;
	double *U_dev1;
	double *U_old_dev0;
	double *U_old_dev1;
	double *F_dev1;
	double *F_dev0;
	double *temp;

	double *U_host0;
	double *U_host1;
	double *U_old_host0;
	double *U_old_host1;
	double *F_host;
	
	//alloctating memory on host
	U_host0 = (double *) malloc(size/2);
	U_old_host0 = (double *) malloc(size/2);
	U_host1 = (double *) malloc(size/2);
	U_old_host1 = (double *) malloc(size/2);
	F_host = (double *) malloc(size);

	//initializing the arrays
	init(N, delta, U_host0, U_old_host0, U_host1, U_old_host1, F_host);
	
	//allocating memory on device0
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipMalloc((void**) &U_dev0,size/2));
	checkCudaErrors(hipMalloc((void**) &U_old_dev0,size/2));
	checkCudaErrors(hipMalloc((void**) &F_dev0, size));
	//allocating memory on device1
	hipSetDevice(1);
	checkCudaErrors(hipMalloc((void**) &U_dev1,size/2));
	checkCudaErrors(hipMalloc((void**) &U_old_dev1,size/2));
	checkCudaErrors(hipMalloc((void**) &F_dev1, size));

	//copying memory from CPU to GPU
	checkCudaErrors(hipMemcpy(U_dev1, U_host1, size/2, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(U_old_dev1, U_old_host1, size/2, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(F_dev1, F_host, size, hipMemcpyHostToDevice));

	hipSetDevice(0);
	checkCudaErrors(hipMemcpy(U_dev0, U_host0, size/2, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(U_old_dev0, U_old_host0, size/2, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(F_dev0, F_host, size, hipMemcpyHostToDevice));
	
	int h;
	for(h = 0; h < k; h++) {
		hipSetDevice(0);
		hipDeviceEnablePeerAccess(1,0);
		jacobi<<<DimGrid, DimBlock>>>(temp_N, temp_N/2, delta2, U_dev0, U_old_dev0, U_dev1, F_dev0,0);
		hipSetDevice(1);
		hipDeviceEnablePeerAccess(0,0);
		jacobi<<<DimGrid, DimBlock>>>(temp_N, temp_N/2, delta2, U_dev1, U_old_dev1,  U_dev0, F_dev1,1);
		//swapping pointers
        	temp = U_dev0;
                U_dev0 = U_old_dev0;
                U_old_dev0 = temp;
		temp = U_dev1;
		U_dev1 = U_old_dev1;
		U_old_dev1 = temp;
	}
	checkCudaErrors(hipMemcpy(U_host1, U_dev1, size/2, hipMemcpyDeviceToHost));
	hipSetDevice(0);
	checkCudaErrors(hipMemcpy(U_host0, U_dev0, size/2, hipMemcpyDeviceToHost));
	print_matrix(N, U_host0,U_host1);
	//freeing the memory in the end
	free(U_host0);
	free(U_old_host0);
	free(U_host1);
	free(U_old_host1);
	free(F_host);
	checkCudaErrors(hipFree(U_dev0));
	checkCudaErrors(hipFree(U_old_dev0));
	checkCudaErrors(hipFree(F_dev0));
	hipSetDevice(1);
	checkCudaErrors(hipFree(U_dev1));
	checkCudaErrors(hipFree(U_old_dev1));
	checkCudaErrors(hipFree(F_dev1));
	return 0;
}
