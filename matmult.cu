#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

extern "C" {
#include <cblas.h>

#define BLOCK_SIZE 32

    void matmult_nat(int m, int n, int k, double *A, double *B, double *C)
    {
        int i, j, h;
        for (i = 0; i < m; i++)
        {
            for (j = 0; j < n; j++)
            {
                C[n * i + j] = 0.0;
                for (h = 0; h < k; h++)
                {
                    C[n * i + j] += A[k * i + h] * B[n * h + j];
                }
            }
        }
    }

    void matmult_mnk(int m, int n, int k, double *A, double *B, double *C)
    {
        int i, j, h;
        for (i = 0; i < m * n; i++)
        {
            C[i] = 0.0;
        }
        for (i = 0; i < m; i++)
        {
            for (j = 0; j < n; j++)
            {
                for (h = 0; h < k; h++)
                {
                    C[n * i + j] += A[k * i + h] * B[n * h + j];
                }
            }
        }
    }

    void matmult_nmk(int m, int n, int k, double *A, double *B, double *C)
    {
        int i, j, h;
        for (i = 0; i < m * n; i++)
        {
            C[i] = 0.0;
        }
        for (j = 0; j < n; j++)
        {
            for (i = 0; i < m; i++)
            {
                for (h = 0; h < k; h++)
                {
                    C[n * i + j] += A[k * i + h] * B[n * h + j];
                }
            }
        }
    }

    void matmult_nkm(int m, int n, int k, double *A, double *B, double *C)
    {
        int i, j, h;
        for (i = 0; i < m * n; i++)
        {
            C[i] = 0.0;
        }
        for (j = 0; j < n; j++)
        {
            for (h = 0; h < k; h++)
            {
                for (i = 0; i < m; i++)
                {
                    C[n * i + j] += A[k * i + h] * B[n * h + j];
                }
            }
        }
    }

    void matmult_kmn(int m, int n, int k, double *A, double *B, double *C)
    {
        int i, j, h;
        for (i = 0; i < m * n; i++)
        {
            C[i] = 0.0;
        }
        for (h = 0; h < k; h++)
        {
            for (i = 0; i < m; i++)
            {
                for (j = 0; j < n; j++)
                {
                    C[n * i + j] += A[k * i + h] * B[n * h + j];
                }
            }
        }
    }

    void matmult_knm(int m, int n, int k, double *A, double *B, double *C)
    {
        int i, j, h;
        for (i = 0; i < m * n; i++)
        {
            C[i] = 0.0;
        }
        for (h = 0; h < k; h++)
        {
            for (j = 0; j < n; j++)
            {
                for (i = 0; i < m; i++)
                {
                    C[n * i + j] += A[k * i + h] * B[n * h + j];
                }
            }
        }
    }

    void matmult_mkn(int m, int n, int k, double *A, double *B, double *C)
    {
        int i, j, h;
        for (i = 0; i < m * n; i++)
        {
            C[i] = 0.0;
        }
        for (i = 0; i < m; i++)
        {
            for (h = 0; h < k; h++)
            {
                for (j = 0; j < n; j++)
                {
                    C[n * i + j] += A[k * i + h] * B[n * h + j];
                }
            }
        }
    }

    void matmult_lib(int m, int n, int k, double *A, double *B, double *C)
    {
        cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1.0, A, k, B, n, 0.0, C, n);
    }

    __global__ void matmult_gpu1_kernel(int m, int n, int k, double *A, double *B, double *C)
    {
        int col = threadIdx.x + blockIdx.x * blockDim.x;
        int row = threadIdx.y + blockIdx.y * blockDim.y;
        if (row < m && col < n)
        {
            double sum = 0.0;
            int i;
            for (i = 0; i < k; i++)
            {
                sum += A[k * row + i] * B[n * i + col];
            }
            C[n * row + col] = sum;
        }
    }

    void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C)
    {
        double *device_a, *device_b, *device_c;
        checkCudaErrors(hipMalloc((void **) &device_a, m * k * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_b, k * n * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_c, m * n * sizeof(double)));

        checkCudaErrors(hipMemcpy(device_a, A, m * k * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(device_b, B, k * n * sizeof(double), hipMemcpyHostToDevice));

        dim3 DimGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE);

        matmult_gpu1_kernel <<< DimGrid, DimBlock >>> (m, n, k, device_a, device_b, device_c);

        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipMemcpy(C, device_c, m * n * sizeof(double), hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(device_a));
        checkCudaErrors(hipFree(device_b));
        checkCudaErrors(hipFree(device_c));
    }

    __global__ void matmult_gpu2_kernel(int m, int n, int k, double *A, double *B, double *C)
    {
        int col = threadIdx.x + blockIdx.x * blockDim.x;
        int row = threadIdx.y + blockIdx.y * blockDim.y * 2;

        double sum = 0.0;
        int i;
        if (row < m && col < n)
        {
            for (i = 0; i < k; i++)
            {
                sum += A[k * row + i] * B[n * i + col];
            }
            C[n * row + col] = sum;
        }
        row += 1;
        if (row < m && col < n)
        {
	        sum = 0.0;
	        for (i = 0; i < k; i++)
	        {
	        	sum += A[k * row + i] * B[n * i + col];
	        }
	        C[n * row + col] = sum;
	    }
    }

    void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C)
    {
        double *device_a, *device_b, *device_c;
        checkCudaErrors(hipMalloc((void **) &device_a, m * k * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_b, k * n * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_c, m * n * sizeof(double)));

        checkCudaErrors(hipMemcpy(device_a, A, m * k * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(device_b, B, k * n * sizeof(double), hipMemcpyHostToDevice));

        dim3 DimGrid((n / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE, (m / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE);

        matmult_gpu2_kernel <<< DimGrid, DimBlock >>> (m, n, k, device_a, device_b, device_c);

        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipGetLastError());

        checkCudaErrors(hipMemcpy(C, device_c, m * n * sizeof(double), hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(device_a));
        checkCudaErrors(hipFree(device_b));
        checkCudaErrors(hipFree(device_c));
    }

    __global__ void matmult_gpu3_kernel(int m, int n, int k, double *A, double *B, double *C)
    {

    }

    void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C)
    {
        double *device_a, *device_b, *device_c;
        checkCudaErrors(hipMalloc((void **) &device_a, m * k * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_b, k * n * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_c, m * n * sizeof(double)));

        checkCudaErrors(hipMemcpy(device_a, A, m * k * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(device_b, B, k * n * sizeof(double), hipMemcpyHostToDevice));

        dim3 DimGrid((n / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE, (m / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE);

        matmult_gpu3_kernel <<< DimGrid, DimBlock >>> (m, n, k, device_a, device_b, device_c);

        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipGetLastError());

        checkCudaErrors(hipMemcpy(C, device_c, m * n * sizeof(double), hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(device_a));
        checkCudaErrors(hipFree(device_b));
        checkCudaErrors(hipFree(device_c));
    }

    __global__ void matmult_gpu4_kernel(int m, int n, int k, double *A, double *B, double *C)
    {
    	int col = threadIdx.x + blockIdx.x * blockDim.x;
        int row = threadIdx.y + blockIdx.y * blockDim.y;

        __shared__ double A_s[BLOCK_SIZE][BLOCK_SIZE + 1];

        if (row < m && col < n) {
	        double sum = 0.0;
		     for (int i = 0; i < k; i += BLOCK_SIZE) {
		        A_s[threadIdx.y][threadIdx.x] = A[i * BLOCK_SIZE * blockIdx.y + i];//A[blockIdx.y * k + i + blockIdx.x];
		        __syncthreads();
		        for (int j = 0; j < BLOCK_SIZE; j++) {
		           sum += A_s[threadIdx.y][j] * B[n * j + col];
		        }
		        C[n * row + col] = sum;
		     }
    	}
    }

    void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C)
    {
        double *device_a, *device_b, *device_c;
        checkCudaErrors(hipMalloc((void **) &device_a, m * k * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_b, k * n * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_c, m * n * sizeof(double)));

        checkCudaErrors(hipMemcpy(device_a, A, m * k * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(device_b, B, k * n * sizeof(double), hipMemcpyHostToDevice));

        dim3 DimGrid((n / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE, (m / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE);

        matmult_gpu4_kernel <<< DimGrid, DimBlock >>> (m, n, k, device_a, device_b, device_c);

        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipMemcpy(C, device_c, m * n * sizeof(double), hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(device_a));
        checkCudaErrors(hipFree(device_b));
        checkCudaErrors(hipFree(device_c));
    }

    __global__ void matmult_gpu5_kernel(int m, int n, int k, double *A, double *B, double *C)
    {

    }

    void matmult_gpu5(int m, int n, int k, double *A, double *B, double *C)
    {
        double *device_a, *device_b, *device_c;
        checkCudaErrors(hipMalloc((void **) &device_a, m * k * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_b, k * n * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_c, m * n * sizeof(double)));

        checkCudaErrors(hipMemcpy(device_a, A, m * k * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(device_b, B, k * n * sizeof(double), hipMemcpyHostToDevice));

        dim3 DimGrid((n / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE, (m / 2 + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE);

        matmult_gpu5_kernel <<< DimGrid, DimBlock >>> (m, n, k, device_a, device_b, device_c);

        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipMemcpy(C, device_c, m * n * sizeof(double), hipMemcpyDeviceToHost));

        checkCudaErrors(hipFree(device_a));
        checkCudaErrors(hipFree(device_b));
        checkCudaErrors(hipFree(device_c));
    }

    void matmult_gpulib(int m, int n, int k, double *A, double *B, double *C)
    {

        double *device_a, *device_b, *device_c, *trans_C;
        checkCudaErrors(hipMalloc((void **) &device_a, m * k * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_b, k * n * sizeof(double)));
        checkCudaErrors(hipMalloc((void **) &device_c, m * n * sizeof(double)));
        trans_C = (double *) malloc(m * n * sizeof(double));

        checkCudaErrors(hipMemcpy(device_a, A, m * k * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(device_b, B, k * n * sizeof(double), hipMemcpyHostToDevice));

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        const double alpha = 1.0, beta = 0.0;
        hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &alpha, device_a, k, device_b, n, &beta, device_c, m);
        hipblasDestroy(handle);

        checkCudaErrors(hipMemcpy(trans_C, device_c, m * n * sizeof(double), hipMemcpyDeviceToHost));

        int row, col;
        for (row = 0; row < m; row++)
        {
            for (col = 0; col < n; col++)
            {
                C[n * row + col] = trans_C[m * col + row];
            }
        }

        checkCudaErrors(hipFree(device_a));
        checkCudaErrors(hipFree(device_b));
        checkCudaErrors(hipFree(device_c));
    }
}
